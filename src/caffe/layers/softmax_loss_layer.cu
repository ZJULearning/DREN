#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(max(prob_data[n * dim + label_value * spatial_dim + s],
                      Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  if(cross_entropy_){
	  // The forward pass computes the softmax prob values.
	  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
	  const Dtype* prob_data = prob_.cpu_data();
	  const Dtype* label = bottom[1]->cpu_data();
	  int dim = prob_.count() / outer_num_;
	  int count = 0;
	  Dtype loss = 0;
	  for (int i = 0; i < outer_num_; ++i) {
		  
		for (int j = 0; j < inner_num_; j++) {
		  const int label_value = static_cast<int>(label[i * inner_num_ + j]);
		  if (has_ignore_label_ && label_value == ignore_label_) {
			continue;
		  }
		  DCHECK_GE(label_value, 0);
		  DCHECK_LT(label_value, prob_.shape(softmax_axis_));
		 /********************************************************************/
		 //add cross_entropy mode 
		 //author: Junying Li data:2017/1/3
		 /********************************************************************/ 
		  if(cross_entropy_){
			  for (int k = 0; k < prob_.shape(softmax_axis_); k++) {
				  loss -= (k == label_value)?log(std::max(prob_data[i * dim + k * inner_num_ + j],Dtype(FLT_MIN))):
					log(std::max(1-prob_data[i * dim + k * inner_num_ + j],Dtype(FLT_MIN)));
				}
		  }
		  else{
			  loss -= log(std::max(prob_data[i * dim + label_value * inner_num_ + j],
								   Dtype(FLT_MIN)));
		  }
		  ++count;
		  /********************************************************************/ 
		}
		
	  }
	  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_, count);
	  if (top.size() == 2) {
		top[1]->ShareData(prob_);
	  }
  }
  else{
	  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
	  const Dtype* prob_data = prob_.gpu_data();
	  const Dtype* label = bottom[1]->gpu_data();
	  const int dim = prob_.count() / outer_num_;
	  const int nthreads = outer_num_ * inner_num_;
	  // Since this memory is not used for anything until it is overwritten
	  // on the backward pass, we use it here to avoid having to allocate new GPU
	  // memory to accumulate intermediate results in the kernel.
	  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
	  // Similarly, this memory is never used elsewhere, and thus we can use it
	  // to avoid having to allocate additional GPU memory.
	  Dtype* counts = prob_.mutable_gpu_diff();
	  // NOLINT_NEXT_LINE(whitespace/operators)
	  SoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
		  CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
		  outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);
	  Dtype loss;
	  caffe_gpu_asum(nthreads, loss_data, &loss);
	  Dtype valid_count = -1;
	  // Only launch another CUDA kernel if we actually need the count of valid
	  // outputs.
	  if (normalization_ == LossParameter_NormalizationMode_VALID &&
		  has_ignore_label_) {
		caffe_gpu_asum(nthreads, counts, &valid_count);
	  }
	  top[0]->mutable_cpu_data()[0] = loss / get_normalizer(normalization_,
															valid_count);
	  if (top.size() == 2) {
		top[1]->ShareData(prob_);
	  }
	}
}

template <typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts) {

  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    const int s = index % spatial_dim;
    const int label_value = static_cast<int>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if(cross_entropy_){
	  if (propagate_down[1]) {
		LOG(FATAL) << this->type()
				   << " Layer cannot backpropagate to label inputs.";
	  }
	  if (propagate_down[0]) {
		Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
		const Dtype* prob_data = prob_.cpu_data();
		caffe_copy(prob_.count(), prob_data, bottom_diff);
		const Dtype* label = bottom[1]->cpu_data();
		int dim = prob_.count() / outer_num_;
		int count = 0;
		for (int i = 0; i < outer_num_; ++i) {
		  for (int j = 0; j < inner_num_; ++j) {
			const int label_value = static_cast<int>(label[i * inner_num_ + j]);
			if (has_ignore_label_ && label_value == ignore_label_) {
			  for (int c = 0; c < bottom[0]->shape(softmax_axis_); ++c) {
				bottom_diff[i * dim + c * inner_num_ + j] = 0;
			  }
			} else {
				/********************************************************************/
				//add cross_entropy mode 
				//author: Junying Li data:2017/1/3
				/********************************************************************/
			  if(cross_entropy_){
				  bottom_diff[i * dim + label_value * inner_num_ + j] -= 1;
			  }
			  else{
				  bottom_diff[i * dim + label_value * inner_num_ + j] -= 1;
			  }
			  ++count;
			  /********************************************************************/
			}
		  }
		}
		// Scale gradient
		Dtype loss_weight = top[0]->cpu_diff()[0] /
							get_normalizer(normalization_, count);
		caffe_scal(prob_.count(), loss_weight, bottom_diff);
	  }
  }
  else{
	  
	  if (propagate_down[1]) {
		LOG(FATAL) << this->type()
				   << " Layer cannot backpropagate to label inputs.";
	  }
	  if (propagate_down[0]) {
		Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
		const Dtype* prob_data = prob_.gpu_data();
		const Dtype* top_data = top[0]->gpu_data();
		caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
		const Dtype* label = bottom[1]->gpu_data();
		const int dim = prob_.count() / outer_num_;
		const int nthreads = outer_num_ * inner_num_;
		// Since this memory is never used for anything else,
		// we use to to avoid allocating new GPU memory.
		Dtype* counts = prob_.mutable_gpu_diff();
		// NOLINT_NEXT_LINE(whitespace/operators)
		SoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
			CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
			outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts);

		Dtype valid_count = -1;
		// Only launch another CUDA kernel if we actually need the count of valid
		// outputs.
		if (normalization_ == LossParameter_NormalizationMode_VALID &&
			has_ignore_label_) {
		  caffe_gpu_asum(nthreads, counts, &valid_count);
		}
		const Dtype loss_weight = top[0]->cpu_diff()[0] /
								  get_normalizer(normalization_, valid_count);
		caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
	  }
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
